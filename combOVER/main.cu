#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include "include/gpuerrchk.h"
#include "include/hide_harppi.h"
#include "include/mcmc.h"

int main(int argc, char *argv[]) {
    mcmc_parameters p(argv[1]);
    
    // Allocate some memory on the GPU
    float3 *d_ks;
    gpuErrchk(hipMalloc((void **)&d_ks, p.num_data*sizeof(float3)));
    double *d_Bk;
    gpuErrchk(hipMalloc((void **)&d_Bk, p.num_data*sizeof(double)));
    
    // Initialize the mcmc object
    mcmc combFit(p, d_ks, d_Bk);
    
    // Run the MCMC chain
    combFit.run_chain(d_ks, d_Bk);
    
    // Free the GPU memory
    gpuErrchk(hipFree(d_ks));
    gpuErrchk(hipFree(d_Bk));
    
    return 0;
}
