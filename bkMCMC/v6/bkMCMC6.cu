#include "hip/hip_runtime.h"
/* bkMCMC5.cu
 * David W. Pearson
 * July 7, 2017
 * 
 * This program makes use of the bkmcmc4.h header file in order to run the MCMC chains for fitting the
 * linear bispectrum model to input data. All of the needed functions are defined in the header file.
 * This program is just to set up the device pointers, and call the functions from the header in the 
 * appropriate order..
 */

#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <gpuerrchk.h>
#include "bkmcmc6.h"
#include "hide_harppi.h"
#include "make_spline.h"

int main(int argc, char *argv[]) {
    // Use HARPPI hidden in an object file to parse parameters
    mcmc_parameters p(argv[1]);
    
    // Generate cubic splines of the input BAO and NW power spectra
    std::vector<float4> Pk = make_spline(p.input_bao_power);
    
    // Copy the splines to the allocated GPU memory
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_Pk), Pk.data(), 128*sizeof(float4)));
    
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_wi), &w_i[0], 32*sizeof(float)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_xi), &x_i[0], 32*sizeof(float)));
    
    // Declare a pointer for the integration workspace and allocate memory on the GPU
    double *d_Bk;
    float3 *d_ks;
    
    gpuErrchk(hipMalloc((void **)&d_Bk, 314*sizeof(double)));
    gpuErrchk(hipMalloc((void **)&d_ks, 314*sizeof(float3)));
    
    // Initialize bkmcmc object
    bkmcmc bk_fit(p.data_file, p.cov_file, p.start_params, p.var_i, d_ks, d_Bk);
    
    // Check that the initialization worked
    bk_fit.check_init();
    
    // Set any limits on the parameters
    bk_fit.set_param_limits(p.limit_params, p.min, p.max);
    
    // Run the MCMC chain
    bk_fit.run_chain(p.num_draws, p.reals_file, d_ks, d_Bk, p.new_chain);
    
    // Free device pointers
    gpuErrchk(hipFree(d_Bk));
    gpuErrchk(hipFree(d_ks));
    
    return 0;
}
